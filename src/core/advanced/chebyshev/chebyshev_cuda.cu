#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

namespace reality_stone::advanced {

// 올바른 tanh 체비셰프 계수
__constant__ float d_tanh_coeffs[16] = {
    0.0f,           // T_0
    0.7615941560f,  // T_1
    0.0f,           // T_2
    -0.2299769450f, // T_3
    0.0f,           // T_4
    0.0469100770f,  // T_5
    0.0f,           // T_6
    -0.0081518632f, // T_7
    0.0f,           // T_8
    0.0013182420f,  // T_9
    0.0f,           // T_10
    -0.0002049517f, // T_11
    0.0f,           // T_12
    0.0000308396f,  // T_13
    0.0f,           // T_14
    -0.0000045292f  // T_15
};

// 동적 곡률 제한 해제 함수들
__device__ float get_dynamic_limit(float curvature) {
    // 곡률에 따른 동적 제한값 계산
    float base_limit = 0.999f;
    float curvature_factor = fminf(curvature, 10.0f);  // 최대 10까지
    return base_limit + (1.0f - base_limit) * (curvature_factor / 10.0f);
}

__device__ float get_dynamic_scale_limit(float curvature) {
    // 스케일링 제한값을 곡률에 따라 동적 조정
    if (curvature <= 1.0f) return 3.0f;      // 기본값
    if (curvature <= 5.0f) return 5.0f;      // 중간 곡률
    return 10.0f + curvature * 0.5f;         // 높은 곡률: 더 넓은 범위
}

__device__ float chebyshev_polynomial(float x, int n) {
    // Clenshaw 재귀로 효율적 계산
    if (n == 0) return 1.0f;
    if (n == 1) return x;
    // T_n(x) = 2x*T_{n-1}(x) - T_{n-2}(x)
    float T_prev2 = 1.0f;
    float T_prev1 = x;
    for (int i = 2; i <= n; ++i) {
        float T_curr = 2.0f * x * T_prev1 - T_prev2;
        T_prev2 = T_prev1;
        T_prev1 = T_curr;
    }
    
    return T_prev1;
}

__global__ void chebyshev_approximation_kernel(
    const float* x,
    float* result,
    int batch_size,
    int dim,
    int order,
    float curvature
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * dim) return;
    
    float x_val = x[idx];
    float sqrt_c = sqrtf(curvature);
    
    // 동적 곡률 제한 해제: 곡률에 따른 스케일링 범위 조정
    const float L = get_dynamic_scale_limit(curvature);  // 동적 스케일 범위
    float scaled_x = sqrt_c * x_val / L;
    
    // 범위 확인 (동적 제한 적용)
    if (fabsf(sqrt_c * x_val) > L) {
        // 범위 밖: tanh(큰값) ≈ ±1
        result[idx] = (x_val > 0) ? 0.99999f : -0.99999f;
        return;
    }
    
    // 동적 제한으로 클리핑 범위 확장
    float dynamic_limit = get_dynamic_limit(curvature);
    scaled_x = fmaxf(-dynamic_limit, fminf(dynamic_limit, scaled_x));
    
    float sum = 0.0f;
    int max_order = min(order, 15);  // 계수 배열 크기 제한
    for (int n = 0; n <= max_order; ++n) {
        float coeff = d_tanh_coeffs[n];
        if (fabsf(coeff) < 1e-8f) continue;  // 0인 계수 스킵
        float T_n = chebyshev_polynomial(scaled_x, n);
        sum += coeff * T_n;
    }
    result[idx] = sum;
}

__global__ void chebyshev_distance_kernel(
    const float* x,
    const float* y,
    float* result,
    int batch_size,
    int dim,
    float curvature
) {
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx >= batch_size) return;
    // 체비셰프 거리 (max norm) 계산
    float max_diff = 0.0f;
    for (int d = 0; d < dim; ++d) {
        float diff = fabsf(x[batch_idx * dim + d] - y[batch_idx * dim + d]);
        max_diff = fmaxf(max_diff, diff);
    }
    // 하이퍼볼릭 공간 변환: d_H = (1/√c) * atanh(√c * d_cheb)
    float sqrt_c = sqrtf(curvature);
    
    // 동적 곡률 제한 해제: 거리 계산시 제한값 확장
    float distance_limit = get_dynamic_limit(curvature);
    float scaled_dist = fmaxf(0.0f, fminf(distance_limit, sqrt_c * max_diff));
    
    result[batch_idx] = (1.0f / sqrt_c) * atanhf(scaled_dist);
}

__global__ void fast_chebyshev_transform_kernel(
    const float* values,
    float* coeffs,
    int n
) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= n) return;
    
    float coeff = 0.0f;
    
    // Type-I DCT 구현
    for (int j = 0; j < n; ++j) {
        float x_j = cosf(M_PI * j / (n - 1));  // 체비셰프 노드
        float T_k = chebyshev_polynomial(x_j, k);
        coeff += values[j] * T_k;
    }
    
    // 정규화
    float norm_factor = (k == 0 || k == n - 1) ? 1.0f : 2.0f;
    coeffs[k] = norm_factor * coeff / n;
}

torch::Tensor chebyshev_approximation_cuda(
    const torch::Tensor& x,
    int order,
    float curvature
) {
    auto batch_size = x.size(0);
    auto dim = x.size(-1);
    auto result = torch::zeros_like(x);
    
    const int threads = 256;
    const int blocks = (batch_size * dim + threads - 1) / threads;
    
    chebyshev_approximation_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        result.data_ptr<float>(),
        batch_size,
        dim,
        order,
        curvature
    );
    
    hipDeviceSynchronize();
    return result;
}

torch::Tensor chebyshev_distance_cuda(
    const torch::Tensor& x,
    const torch::Tensor& y,
    float curvature
) {
    auto batch_size = x.size(0);
    auto dim = x.size(-1);
    auto result = torch::zeros({batch_size}, x.options());
    
    const int threads = 256;
    const int blocks = (batch_size + threads - 1) / threads;
    
    chebyshev_distance_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        result.data_ptr<float>(),
        batch_size,
        dim,
        curvature
    );
    
    hipDeviceSynchronize();
    return result;
}

torch::Tensor fast_chebyshev_transform_cuda(
    const torch::Tensor& values
) {
    auto n = values.size(-1);
    auto coeffs = torch::zeros_like(values);
    
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    
    fast_chebyshev_transform_kernel<<<blocks, threads>>>(
        values.data_ptr<float>(),
        coeffs.data_ptr<float>(),
        n
    );
    
    hipDeviceSynchronize();
    return coeffs;
}

__global__ void inverse_chebyshev_transform_kernel(
    const float* coeffs,
    const float* eval_points,
    float* result,
    int batch_size,
    int order,
    int eval_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * eval_size) return;
    
    int batch_idx = idx / eval_size;
    int eval_idx = idx % eval_size;
    
    float x = eval_points[eval_idx];
    x = fmaxf(-0.999f, fminf(0.999f, x));  // clamp
    
    float sum = 0.0f;
    for (int k = 0; k <= order; ++k) {
        float T_k = chebyshev_polynomial(x, k);
        sum += coeffs[batch_idx * (order + 1) + k] * T_k;
    }
    
    result[idx] = sum;
}

__global__ void chebyshev_derivative_kernel(
    const float* coeffs,
    float* d_coeffs,
    int batch_size,
    int n
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * (n - 1)) return;
    
    int batch_idx = idx / (n - 1);
    int k = idx % (n - 1);
    
    float d_coeff = 0.0f;
    
    // 체비셰프 미분 점화식
    if (k == n - 2) {
        d_coeff = 2.0f * (k + 1) * coeffs[batch_idx * n + (k + 1)];
    } else {
        d_coeff = d_coeffs[batch_idx * (n - 1) + (k + 2)] + 2.0f * (k + 1) * coeffs[batch_idx * n + (k + 1)];
    }
    
    d_coeffs[idx] = d_coeff;
}

__global__ void chebyshev_integral_kernel(
    const float* coeffs,
    float* int_coeffs,
    float constant,
    int batch_size,
    int n
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * (n + 1)) return;
    
    int batch_idx = idx / (n + 1);
    int k = idx % (n + 1);
    
    if (k == 0) {
        int_coeffs[idx] = constant;
    } else if (k == 1 && n > 2) {
        int_coeffs[idx] = coeffs[batch_idx * n + 0] - coeffs[batch_idx * n + 2] / 4.0f;
    } else if (k == 1) {
        int_coeffs[idx] = coeffs[batch_idx * n + 0];
    } else if (k == n) {
        int_coeffs[idx] = coeffs[batch_idx * n + (n - 1)] / (2.0f * k);
    } else {
        int_coeffs[idx] = (coeffs[batch_idx * n + (k - 1)] - coeffs[batch_idx * n + (k + 1)]) / (2.0f * k);
    }
}

torch::Tensor inverse_chebyshev_transform_cuda(
    const torch::Tensor& coeffs,
    const torch::Tensor& eval_points
) {
    auto batch_size = coeffs.size(0);
    auto order = coeffs.size(-1) - 1;
    auto eval_size = eval_points.size(-1);
    auto result = torch::zeros({batch_size, eval_size}, coeffs.options());
    
    const int threads = 256;
    const int blocks = (batch_size * eval_size + threads - 1) / threads;
    
    inverse_chebyshev_transform_kernel<<<blocks, threads>>>(
        coeffs.data_ptr<float>(),
        eval_points.data_ptr<float>(),
        result.data_ptr<float>(),
        static_cast<int>(batch_size),
        static_cast<int>(order),
        static_cast<int>(eval_size)
    );
    
    hipDeviceSynchronize();
    return result;
}

torch::Tensor chebyshev_derivative_cuda(
    const torch::Tensor& coeffs
) {
    auto batch_size = coeffs.size(0);
    auto n = coeffs.size(-1);
    if (n <= 1) return torch::zeros({batch_size, 1}, coeffs.options());
    auto d_coeffs = torch::zeros({batch_size, n - 1}, coeffs.options());
    const int threads = 256;
    const int blocks = (batch_size * (n - 1) + threads - 1) / threads;
    // 역순으로 계산 (점화식 때문)
    for (int k = n - 2; k >= 0; --k) {
        dim3 grid((batch_size + threads - 1) / threads);
        chebyshev_derivative_kernel<<<grid, threads>>>(
            coeffs.data_ptr<float>(),
            d_coeffs.data_ptr<float>(),
            static_cast<int>(batch_size),
            static_cast<int>(n)
        );
    }
    
    hipDeviceSynchronize();
    return d_coeffs;
}

torch::Tensor chebyshev_integral_cuda(
    const torch::Tensor& coeffs,
    float constant
) {
    auto batch_size = coeffs.size(0);
    auto n = coeffs.size(-1);
    auto int_coeffs = torch::zeros({batch_size, n + 1}, coeffs.options());
    
    const int threads = 256;
    const int blocks = (batch_size * (n + 1) + threads - 1) / threads;
    
    chebyshev_integral_kernel<<<blocks, threads>>>(
        coeffs.data_ptr<float>(),
        int_coeffs.data_ptr<float>(),
        constant,
        static_cast<int>(batch_size),
        static_cast<int>(n)
    );
    
    hipDeviceSynchronize();
    return int_coeffs;
}

}