#include "hip/hip_runtime.h"
#ifdef WITH_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif
#include <torch/extension.h>
#include <utils/cuda_utils.h>
#include <utils/numeric.h>
#include <config/constant.h>
#include <layers/poincare_ball.h>

namespace utils = reality_stone::utils;
namespace config = reality_stone::config;

namespace reality_stone::layers {
    template <typename scalar_t>
    __global__ void poincare_ball_backward_kernel(
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ u,
        const scalar_t* __restrict__ v,
        scalar_t* __restrict__ grad_u,
        scalar_t* __restrict__ grad_v,
        float c, float t, int B, int D
    ) {
        int bid = blockIdx.x;
        int tid = threadIdx.x;
        int blockSize = blockDim.x;

        if (bid >= B) return;

        const scalar_t* u_bid = u + bid * D;
        const scalar_t* v_bid = v + bid * D;
        scalar_t* grad_u_bid = grad_u + bid * D;
        scalar_t* grad_v_bid = grad_v + bid * D;
        const scalar_t* grad_out_bid = grad_output + bid * D;
        float c2 = c * c;
        for (int d = tid; d < D; d += blockSize) {
            float u_val = u_bid[d];
            float v_val = v_bid[d];
            float grad_out_val = grad_out_bid[d];
            float jacob_u = 1.0f - t;
            float jacob_v = t;
            grad_u_bid[d] = grad_out_val * jacob_u;
            grad_v_bid[d] = grad_out_val * jacob_v;
        }
    }

    std::tuple<torch::Tensor, torch::Tensor> poincare_ball_backward_cuda(
        torch::Tensor grad_output,
        torch::Tensor u,
        torch::Tensor v,
        float c,
        float t
    ) {
        TORCH_CHECK(grad_output.device().is_cuda(), "grad_output must be on GPU");
        TORCH_CHECK(u.device().is_cuda(), "u must be on GPU");
        TORCH_CHECK(v.device().is_cuda(), "v must be on GPU");

        TORCH_CHECK(grad_output.is_contiguous(), "grad_output must be contiguous");
        TORCH_CHECK(u.is_contiguous(), "u must be contiguous");
        TORCH_CHECK(v.is_contiguous(), "v must be contiguous");

        int B = u.size(0), D = u.size(1);
        auto grad_u = torch::zeros_like(u);
        auto grad_v = torch::zeros_like(v);

        int threads = 256;
        int blocks = B;

        AT_DISPATCH_FLOATING_TYPES(u.scalar_type(), "poincare_ball_backward_cuda", [&] {
            poincare_ball_backward_kernel<scalar_t> << <blocks, threads >> > (
                grad_output.data_ptr<scalar_t>(),
                u.data_ptr<scalar_t>(),
                v.data_ptr<scalar_t>(),
                grad_u.data_ptr<scalar_t>(),
                grad_v.data_ptr<scalar_t>(),
                c, t, B, D
                );
            });

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        return std::make_tuple(grad_u, grad_v);
    }
}
