#include "hip/hip_runtime.h"
#ifdef WITH_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif
#include <torch/extension.h>
#include <ops/lorentz.h>
#include <layers/lorentz.h>
#include <utils/cuda_utils.h>
#include <config/constant.h>

namespace ops = reality_stone::ops;
namespace utils = reality_stone::utils;
namespace config = reality_stone::config;

namespace reality_stone::layers {

    template <typename scalar_t>
    __global__ void lorentz_forward_kernel(
        const scalar_t* __restrict__ u,
        const scalar_t* __restrict__ v,
        scalar_t* __restrict__ result,
        int B, int D, float c, float t
    ) {
        int bid = blockIdx.x;
        if (bid >= B) return;

        const scalar_t* u_bid = u + bid * D;
        const scalar_t* v_bid = v + bid * D;
        scalar_t* result_bid = result + bid * D;
        scalar_t inner = u_bid[0] * v_bid[0];
        for (int d = 1; d < D; ++d) {
            inner -= u_bid[d] * v_bid[d];
        }
        inner = fmaxf(-inner, 1.0f + config::Constants::EPS);
        scalar_t dist = acoshf(inner) / sqrtf(c);
        scalar_t v_perp[32];  // 스택에 임시 저장 (제한된 차원 수)
        for (int d = 0; d < D; ++d) {
            v_perp[d] = v_bid[d] + inner * u_bid[d];
        }
        scalar_t v_perp_norm_sq = v_perp[0] * v_perp[0];
        for (int d = 1; d < D; ++d) {
            v_perp_norm_sq -= v_perp[d] * v_perp[d];
        }
        scalar_t v_perp_norm = sqrtf(fmaxf(-v_perp_norm_sq, 1e-8f));
        scalar_t cosh_dist_t = coshf(dist * t);
        scalar_t sinh_dist_t = sinhf(dist * t);
        for (int d = 0; d < D; ++d) {
            result_bid[d] = cosh_dist_t * u_bid[d] + sinh_dist_t * v_perp[d] / v_perp_norm;
        }
    }

    torch::Tensor lorentz_forward_cuda(torch::Tensor u, torch::Tensor v, float c, float t) {
        utils::check_cuda_tensor(u);
        utils::check_cuda_tensor(v);
        int B = u.size(0), D = u.size(1);
        auto result = torch::empty_like(u);
        int threads = 256;
        int blocks = (B + threads - 1) / threads;
        AT_DISPATCH_FLOATING_TYPES(u.scalar_type(), "lorentz_forward_cuda", [&] {
            lorentz_forward_kernel<scalar_t> << <blocks, 1 >> > (
                u.data_ptr<scalar_t>(),
                v.data_ptr<scalar_t>(),
                result.data_ptr<scalar_t>(),
                B, D, c, t
                );
            });
        utils::check_cuda_error();
        return result;
    }
}