#include "hip/hip_runtime.h"
// hyper_butterfly/csrc/transforms/butterfly/forward.cu
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "butterfly_forward.h"
#include "../../utils/common_defs.h"

using namespace std;
using namespace torch;

namespace hyper_butterfly {
namespace ops {
namespace butterfly {

// 버터플라이 레이어 순전파 커널
template <typename scalar_t>
__global__ void butterfly_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ params,
    int B, int D, int layer_idx) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int bs = 1 << layer_idx;
    int nb = D / (2 * bs);

    while (idx < B * D) {
        int b = idx / D, f = idx % D;
        int blk = (f / (2 * bs)) % nb,
            loc = f % (2 * bs),
            off = loc % bs;
        bool high = loc >= bs;
        int pi = blk * 2;
        float a = params[pi + 0],
            bb = params[pi + 1];
        int base = b * D + blk * 2 * bs;
        float x1 = input[base + off],
            x2 = input[base + off + bs];
        output[idx] = high
            ? (-bb * x1 + a * x2)
            : (a * x1 + bb * x2);
        idx += stride;
    }
}

// CUDA 버터플라이 레이어 순전파 구현
torch::Tensor butterfly_forward_cuda(
    torch::Tensor input,
    torch::Tensor params,
    int layer_idx,
    int batch_size,
    int dim) {

    CHECK_CUDA_CONTIGUOUS(input);
    CHECK_CUDA_CONTIGUOUS(params);

    auto output = torch::empty_like(input);
    int threads = 256;
    int blocks = (batch_size * dim + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "butterfly_forward_cuda", [&] { butterfly_forward_kernel<scalar_t> << <blocks, threads >> > (
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        params.data_ptr<scalar_t>(),
        batch_size, dim, layer_idx); });

    CUDA_CHECK(hipGetLastError());
    return output;
}

} // namespace butterfly
} // namespace transforms
} // namespace hyper_butterfly

// Python 바인딩을 위한 함수
torch::Tensor butterfly_forward_cuda_export(
    torch::Tensor input,
    torch::Tensor params,
    int layer_idx,
    int batch_size,
    int dim) {
    return hyper_butterfly::ops::butterfly::butterfly_forward_cuda(
        input, params, layer_idx, batch_size, dim);
}